////////////////////////////////////////////////////////////
//Ho Thien Luan -> History Tracking!
// 1. multi_pat_asm_naive_cpu.cu 
// 2. 
//
//
//
////////////////////////////////////////////////////////////
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <assert.h>
#include <time.h>
#include <hip/hip_runtime.h>
//#include "cuPrintf.cu"


#define FILENAME_MAXLEN     256
#define THREAD_BLOCK_EXP   (7)
#define THREAD_BLOCK_SIZE  (1 << THREAD_BLOCK_EXP)

__global__ void ASM_kernel(char *g_input_string, int input_size, int *g_pattern_table, int pattern_length, int no_of_patterns, int *g_matched_result)
{
    int tid  = threadIdx.x ;
    int gbid = blockIdx.y * gridDim.x + blockIdx.x ;
    int start = gbid*THREAD_BLOCK_SIZE + tid;
    int max_pattern_length = pattern_length +1; 
    int result;

  if (start < (input_size-pattern_length+1)) {

    for (int j = 0; j < no_of_patterns; j++) {
      result = 0;
      for (int i = 0; i<pattern_length ; i++) {

        if ((g_input_string[ start + i ] != '\n') & (g_input_string[ start + i ] != g_pattern_table[j*max_pattern_length+i])) {
	    result ++;
	    //g_matched_result[start*no_of_patterns + j] = g_matched_result[start*no_of_patterns + j] + 1;
        }
      }
      g_matched_result[start*no_of_patterns + j] = result;
    }
  }
    //cuPrintf("threadIdx.x = %d \t bit_vector = %d \n", start, bit_vector);
}

////////////////////////////////
void ASM_process_top (char *g_input_string, size_t input_size,  int *g_pattern_table, int pattern_length, int no_of_patterns, int *g_matched_result)
{

    // num_blocks = # of thread blocks to cover input stream
    int num_blocks = (input_size-pattern_length+1)/THREAD_BLOCK_SIZE + 1 ;


        dim3  dimBlock( THREAD_BLOCK_SIZE, 1 ) ;
        dim3  dimGrid ;

        int p = num_blocks >> 15 ;
        dimGrid.x = num_blocks ;
        if ( p ){
            dimGrid.x = 1<<15 ;
            dimGrid.y = p+1 ;
        }
    //cudaPrintfInit();////for cuPrintf

    ASM_kernel <<< dimGrid, dimBlock >>>((char*)g_input_string, input_size, (int*) g_pattern_table, pattern_length, no_of_patterns, g_matched_result);

    //cudaPrintfDisplay();////for cuPrintf
    //cudaPrintfEnd();        ////for cuPrintf
}

int main(int argc, char **argv)
{
    char inputFile[FILENAME_MAXLEN];
    char patternFile[FILENAME_MAXLEN];
    strcpy( inputFile, argv[2]) ;
    strcpy( patternFile, argv[1]) ;
    //int k_par = 4;
    int k_par;
    k_par = strtol(argv[3], NULL, 10);
////////////////////////////////////////////////////////////////////////////////////
//Process input patterns
    int pattern_size;
    char *h_pattern = NULL ;
    int len;

    size_t  sizeOfTableEntry ;
    int  sizeOfTableInBytes ; // numOfTableEntry * sizeOfTableEntry
    size_t  sizeOfPatternInBytes ; // no_of_patterns * sizeOfTableEntry

    int max_pattern_length = 0;
    int no_of_patterns = 0;

    FILE* fpattern = fopen( patternFile, "rb");
    assert ( NULL != fpattern ) ;
    // obtain pattern file
    fseek (fpattern , 0 , SEEK_END);
    pattern_size = ftell (fpattern);
    rewind (fpattern);

    // allocate a buffer to contains all patterns
    h_pattern = (char *) malloc (sizeof(char)*pattern_size);
    assert( NULL != h_pattern );

    // copy the file into the buffer
    pattern_size = fread (h_pattern, 1, pattern_size, fpattern);
    fclose(fpattern);
    //printf ("pattern size = %d\n",pattern_size); 
    //printf ("pattern = %s\n",h_pattern); 
    //Processing to get max_pattern_length & no_of_patterns
    len = 0;
    for( int i = 0 ; i < pattern_size ; i++){
        if ( '\n' == h_pattern[i] ){
            if ( (i > 0) && ('\n' != h_pattern[i-1]) ){ // non-empty line
		no_of_patterns = no_of_patterns + 1;	
		if (max_pattern_length < len+1) {max_pattern_length = len+1;}
            }
            len = 0 ;
        }else{
            len++ ;
        }
    }

   // Create pattern_table, pattern_length_table
    sizeOfTableEntry = sizeof(int) ;
    sizeOfPatternInBytes = no_of_patterns * sizeOfTableEntry; // 1-D to store size of each patterns
    sizeOfTableInBytes = no_of_patterns * max_pattern_length; //2-D to store patterns

    int* pattern_table = (int*) malloc( sizeof(int)*sizeOfTableInBytes ) ;
    int* pattern_length_table = (int*) malloc( sizeOfPatternInBytes ) ;

    //Processing to fill pattern_table & pattern_length_table 
    len = 0;
    int no_patterns = 0;
    for( int i = 0 ; i < pattern_size ; i++){
        if ( '\n' == h_pattern[i] ){
            if ( (i > 0) && ('\n' != h_pattern[i-1]) ){ // non-empty line
		pattern_length_table[no_patterns] = len;
		no_patterns = no_patterns + 1;	
            }
            len = 0 ;
        }else{
	    pattern_table[no_patterns*max_pattern_length + len] = h_pattern[i]; 	
            len++ ;
        }
    }
   //Print to pattern_table/pattern_length_table to check
/*
    for (int i = 0; i < no_of_patterns; i++) {
	printf("\npattern no %d has length = %d-> ",i, pattern_length_table[i]);
	for (int j = 0; j < pattern_length_table[i]; j++) {
	    printf("%4d",pattern_table[i*max_pattern_length+j]);
	}
    }
*/
///////////////////////////////////////////////////////////////
//Prepare input string
    int input_size;
    char *h_input_string = NULL ;
    int  *h_matched_result = NULL ;

    //open to read file
    FILE* fpin = fopen( inputFile, "rb");
    assert ( NULL != fpin ) ;
    
    // sets the file position of the stream to the given offset. 
    fseek (fpin , 0 , SEEK_END);
    input_size = ftell (fpin);
    rewind (fpin);
    
    // allocate memory to contain the whole file
    h_input_string = (char *) malloc (sizeof(char)*input_size);
    assert( NULL != h_input_string );

    int size_matched_result = input_size * no_of_patterns;
    h_matched_result = (int *) malloc (sizeof(int)*size_matched_result); // each input has no_of_patterns results
    assert( NULL != h_matched_result );
    memset( h_matched_result, 0, size_matched_result ) ;

    // copy the file into the buffer
    input_size = fread (h_input_string, 1, input_size, fpin);
    fclose(fpin);

    //////////////////
    //printf("\ninput size -> %4d -> \n",input_size);
   // printf("%s\n",h_input_string);
/*
    //AmSM with Naive Method in CPU
    
    struct timespec t_start, t_end;
    double elapsedTime;
    clock_gettime (CLOCK_REALTIME, &t_start);
    //printf ("starttime s = %li, ns = %li\n",t_start.tv_sec, t_start.tv_nsec);

    for(int i = 0; i < input_size-max_pattern_length+1; i++) {
	for (int j = 0; j < no_of_patterns; j++) { 
	h_matched_result[i*no_of_patterns+j] = 0;
	    for (int k = 0; k < pattern_length_table[j]; k++) {
		if ((h_input_string[i+k] != '\n') & (h_input_string[i+k] != pattern_table[j*max_pattern_length+k])) {
		    h_matched_result[i*no_of_patterns+j] = h_matched_result[i*no_of_patterns+j] + 1;
		}
	    }
        }
    }

    clock_gettime(CLOCK_REALTIME, &t_end);
    //printf ("endtime s = %li, ns = %li\n",t_end.tv_sec, t_end.tv_nsec);
    elapsedTime = (t_end.tv_sec*1000+t_end.tv_nsec/1000000)-(t_start.tv_sec*1000+t_start.tv_nsec/1000000);
*/

    //AmSM with Naive Method in GPU
    char *g_input_string;
    //char *g_pattern;
    int *g_matched_result;
    int *g_pattern_table;

    hipMalloc (&g_input_string, sizeof(char)*input_size);
    hipMalloc (&g_matched_result, sizeof(int)*size_matched_result);
    hipMalloc (&g_pattern_table, sizeof(int)*sizeOfTableInBytes);

    hipMemcpy (g_input_string, h_input_string, sizeof(char)*input_size, hipMemcpyHostToDevice );
    hipMemcpy (g_pattern_table, pattern_table, sizeof(int)*sizeOfTableInBytes, hipMemcpyHostToDevice);

    // record time setting
    hipEvent_t start, stop;
    float time;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);


    // step 3: run ASM on GPU           
    ASM_process_top ( g_input_string, input_size, g_pattern_table, (max_pattern_length-1), no_of_patterns, g_matched_result) ;

    // record time setting
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);

    hipMemcpy (h_matched_result, g_matched_result, sizeof(int)*size_matched_result, hipMemcpyDeviceToHost );


    // Print Result
 int total_result = 0;
    for(int i = 0; i < input_size-max_pattern_length+1; i++) {
	for (int j = 0; j < no_of_patterns; j++) { 
	     //printf("Input location %d with pattern %d has Hamming distance = %d\n",i, j, h_matched_result[RESULT_TABLE_MAP(i,j)]);
	     if(h_matched_result[i*no_of_patterns+j] <= k_par) {total_result++;}
	    }
        }
    printf("\n\n\n");
    printf("###########################################################\n");
    printf("#--Multi Fix-Length Patterns Approximate String Matching--#\n");
    printf("#---------------------------------------------------------#\n");
    printf("#----------------Naive Approach in GPU--------------------#\n");
    printf("###########################################################\n");
    printf("#--No of Patterns            |\t\t %10d \t  #\n",no_of_patterns);
    printf("#---------------------------------------------------------#\n");
    printf("#--Pattern Length            |\t\t %10d \t  #\n",max_pattern_length-1);
    printf("#---------------------------------------------------------#\n");
    printf("#--Input Size (bytes)        |\t\t %10d \t  #\n", input_size );
    printf("#---------------------------------------------------------#\n");
    printf("#--Total matched with k = %d  |\t\t %10d \t  #\n", k_par, total_result);
    printf("#---------------------------------------------------------#\n");
    printf("#--Total elapsed time (ms)   |\t\t %10f \t  #\n", time);
    printf("#---------------------------------------------------------#\n");
    printf("#--Throughput Result (Gbps)  |\t\t %10f \t  #\n", (float)(input_size*8)/(time*1000000) );
    printf("###########################################################\n");

     
    free(h_pattern);
    free(h_input_string);
    free(h_matched_result); 
    free(pattern_table);
    free(pattern_length_table);
            
    return 0;
}
