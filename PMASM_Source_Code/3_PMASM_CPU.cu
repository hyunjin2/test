////////////////////////////////////////////////////////////
//Ho Thien Luan -> History Tracking!
// 1. multi_pat_asm_naive_cpu.cu 
// 2. 
//
//
//
////////////////////////////////////////////////////////////

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <assert.h>
#include <time.h>

#define FILENAME_MAXLEN     256

 
int main(int argc, char **argv)
{
    char inputFile[FILENAME_MAXLEN];
    char patternFile[FILENAME_MAXLEN];
    strcpy( inputFile, argv[2]) ;
    strcpy( patternFile, argv[1]) ;
    //int k_par = 4;
    int k_par;
    k_par = strtol(argv[3], NULL, 10);
////////////////////////////////////////////////////////////////////////////////////
//Process input patterns
    int pattern_size;
    char *h_pattern = NULL ;
    int len;

    size_t  sizeOfTableEntry ;
    size_t  sizeOfTableInBytes ; // numOfTableEntry * sizeOfTableEntry
    size_t  sizeOfTableDecodeInBytes ; // numOfTableEntry * sizeOfTableEntry
    size_t  sizeOfPatternInBytes ; // no_of_patterns * sizeOfTableEntry

    int max_pattern_length = 0;
    int no_of_patterns = 0;

    FILE* fpattern = fopen( patternFile, "rb");
    assert ( NULL != fpattern ) ;
    // obtain pattern file
    fseek (fpattern , 0 , SEEK_END);
    pattern_size = ftell (fpattern);
    rewind (fpattern);

    // allocate a buffer to contains all patterns
    h_pattern = (char *) malloc (sizeof(char)*pattern_size);
    assert( NULL != h_pattern );

    // copy the file into the buffer
    pattern_size = fread (h_pattern, 1, pattern_size, fpattern);
    fclose(fpattern);
    //printf ("pattern size = %d\n",pattern_size); 
    //printf ("pattern = %s\n",h_pattern); 
    //Processing to get max_pattern_length & no_of_patterns
    len = 0;
    for( int i = 0 ; i < pattern_size ; i++){
        if ( '\n' == h_pattern[i] ){
            if ( (i > 0) && ('\n' != h_pattern[i-1]) ){ // non-empty line
		no_of_patterns = no_of_patterns + 1;	
		if (max_pattern_length < len+1) {max_pattern_length = len+1;}
            }
            len = 0 ;
        }else{
            len++ ;
        }
    }

   // Create pattern_table, pattern_length_table
    sizeOfTableEntry = sizeof(int) ;
    sizeOfPatternInBytes = no_of_patterns * sizeOfTableEntry; // 1-D to store size of each patterns
    sizeOfTableInBytes = no_of_patterns * max_pattern_length * sizeOfTableEntry; //2-D to store patterns
    sizeOfTableDecodeInBytes = 4 * (max_pattern_length-1) * sizeOfTableEntry; // 1-D to store size of each patterns

    int* pattern_table = (int*) malloc( sizeOfTableInBytes ) ;
    int* pattern_length_table = (int*) malloc( sizeOfPatternInBytes ) ;
    int* h_pattern_decode = (int*) malloc( sizeOfTableDecodeInBytes ) ;

    //Processing to fill pattern_table & pattern_length_table 
    len = 0;
    int no_patterns = 0;
    for( int i = 0 ; i < pattern_size ; i++){
        if ( '\n' == h_pattern[i] ){
            if ( (i > 0) && ('\n' != h_pattern[i-1]) ){ // non-empty line
		pattern_length_table[no_patterns] = len;
		no_patterns = no_patterns + 1;	
            }
            len = 0 ;
        }else{
	    pattern_table[no_patterns*max_pattern_length + len] = h_pattern[i]; 	
            len++ ;
        }
    }
   //Print to pattern_table/pattern_length_table to check
/*
    for (int i = 0; i < no_of_patterns; i++) {
	printf("\npattern no %d has length = %d-> ",i, pattern_length_table[i]);
	for (int j = 0; j < pattern_length_table[i]; j++) {
	    printf("%4d",pattern_table[i*max_pattern_length+j]);
	}
    }
*/
//printf ("\n");
//Preprocessing
unsigned int vector_A = 0;
unsigned int vector_C = 0;
unsigned int vector_G = 0;
unsigned int vector_T = 0;
int b = 4;
int pow_2b = 1 << b;

    for (int i = 0; i< (max_pattern_length-1); i++) {
	vector_A = 0;
	vector_C = 0;
	vector_G = 0;
	vector_T = 0;
	for (int j = 0; j< no_of_patterns; j++) {
	    vector_A = vector_A << b;
	    if (pattern_table[i + max_pattern_length*j] != 65) {vector_A = vector_A + 1;};
	}
	h_pattern_decode[4*i] = vector_A;
		
	for (int j = 0; j< no_of_patterns; j++) {
	    vector_C = vector_C << b;
	    if (pattern_table[i + max_pattern_length*j] != 67) {vector_C = vector_C + 1;};
	}
	h_pattern_decode[4*i+1] = vector_C;

	for (int j = 0; j< no_of_patterns; j++) {
	    vector_G = vector_G << b;
	    if (pattern_table[i + max_pattern_length*j] != 71) {vector_G = vector_G + 1;};
	}
	h_pattern_decode[4*i+2] = vector_G;

	for (int j = 0; j< no_of_patterns; j++) {
	    vector_T = vector_T << b;
	    if (pattern_table[i + max_pattern_length*j] != 84) {vector_T = vector_T + 1;};
	}
	h_pattern_decode[4*i+3] = vector_T;
    }
	
//    for (int i = 0; i < (max_pattern_length-1)*4; i++) {
//	printf("i = %d -> h_pattern_decode = %d\n",i,h_pattern_decode[i]);
//    }
///////////////////////////////////////////////////////////////
//Prepare input string
    int input_size;
    char *h_input_string = NULL ;
    int  *h_matched_result = NULL ;

    //open to read file
    FILE* fpin = fopen( inputFile, "rb");
    assert ( NULL != fpin ) ;
    
    // sets the file position of the stream to the given offset. 
    fseek (fpin , 0 , SEEK_END);
    input_size = ftell (fpin);
    rewind (fpin);
    
    // allocate memory to contain the whole file
    h_input_string = (char *) malloc (sizeof(char)*input_size);
    assert( NULL != h_input_string );

    size_t size_matched_result = sizeOfTableEntry * input_size * no_of_patterns;
    h_matched_result = (int *) malloc (size_matched_result); // each input has no_of_patterns results
    assert( NULL != h_matched_result );
    memset( h_matched_result, 0, size_matched_result ) ;

    // copy the file into the buffer
    input_size = fread (h_input_string, 1, input_size, fpin);
    fclose(fpin);

    //////////////////
    //printf("\ninput size -> %4d -> \n",input_size);
   // printf("%s\n",h_input_string);

    //AmSM with Naive Method in CPU
    
unsigned int vector = 0;
    struct timespec t_start, t_end;
    double elapsedTime;
    clock_gettime (CLOCK_REALTIME, &t_start);
    //printf ("starttime s = %li, ns = %li\n",t_start.tv_sec, t_start.tv_nsec);
    for(int i = 0; i < input_size-max_pattern_length+1; i++) {
	vector = 0;
	for (int k = 0; k < max_pattern_length-1; k++) {
	     if (h_input_string[i+k] == 65) {
		vector = vector + h_pattern_decode[4*k];
	     }
	     else if (h_input_string[i+k] == 67) {
		vector = vector + h_pattern_decode[4*k+1];
	     }
	     else if (h_input_string[i+k] == 71) {
		vector = vector + h_pattern_decode[4*k+2];
	     }
	     else if (h_input_string[i+k] == 84) {
		vector = vector + h_pattern_decode[4*k+3];
	     }
    	//printf("vector = %d, \n",vector);
	}
	for (int j = no_of_patterns-1; j >= 0; j--) { 
	     h_matched_result[i*no_of_patterns+j] = vector % pow_2b;
	     vector = vector >> b;
        }
    }
    clock_gettime(CLOCK_REALTIME, &t_end);
    //printf ("endtime s = %li, ns = %li\n",t_end.tv_sec, t_end.tv_nsec);
    elapsedTime = (t_end.tv_sec*1000+t_end.tv_nsec/1000000)-(t_start.tv_sec*1000+t_start.tv_nsec/1000000);

    // Print Result
 int total_result = 0;
    for(int i = 0; i < input_size-max_pattern_length+1; i++) {
	for (int j = 0; j < no_of_patterns; j++) { 
	     //printf("Input location %d with pattern %d has Hamming distance = %d\n",i, j, h_matched_result[i*no_of_patterns+j]);
	     if(h_matched_result[i*no_of_patterns+j] <= k_par) {total_result++;}
	    }
        }
    printf("\n\n\n");
    printf("###########################################################\n");
    printf("#--Multi Fix-Length Patterns Approximate String Matching--#\n");
    printf("#---------------------------------------------------------#\n");
    printf("#---------------Proposed PMASM Alg. in CPU----------------#\n");
    printf("###########################################################\n");
    printf("#--No of Patterns            |\t\t %10d \t  #\n",no_of_patterns);
    printf("#---------------------------------------------------------#\n");
    printf("#--Pattern Length            |\t\t %10d \t  #\n",max_pattern_length-1);
    printf("#---------------------------------------------------------#\n");
    printf("#--Input Size (bytes)        |\t\t %10d \t  #\n", input_size );
    printf("#---------------------------------------------------------#\n");
    printf("#--Total matched with k = %d  |\t\t %10d \t  #\n", k_par, total_result);
    printf("#---------------------------------------------------------#\n");
    printf("#--Total elapsed time (ms)   |\t\t %10f \t  #\n", elapsedTime);
    printf("#---------------------------------------------------------#\n");
    printf("#--Throughput Result (Gbps)  |\t\t %10f \t  #\n", (float)(input_size*8)/(elapsedTime*1000000) );
    printf("###########################################################\n");


     
    free(h_pattern);
    free(h_input_string);
    free(h_matched_result); 
    free(pattern_table);
    free(pattern_length_table);
    free(h_pattern_decode);
            
    return 0;
}
