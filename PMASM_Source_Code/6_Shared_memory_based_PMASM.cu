////////////////////////////////////////////////////////////
//Ho Thien Luan -> History Tracking!
// 1. multi_pat_asm_naive_cpu.cu 
// 2. 
//
//
//
////////////////////////////////////////////////////////////
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <assert.h>
#include <time.h>
#include <hip/hip_runtime.h>
//#include "cuPrintf.cu"


#define FILENAME_MAXLEN     256
#define THREAD_BLOCK_EXP   (7)
#define THREAD_BLOCK_SIZE  (1 << THREAD_BLOCK_EXP)

__global__ void ASM_kernel(char *g_input_string, int input_size, int *g_pattern_decode, int pattern_length, int no_of_patterns, int b, int *g_matched_result)
{
    int tid  = threadIdx.x ;
    int gbid = blockIdx.y * gridDim.x + blockIdx.x ;
    int start = gbid*THREAD_BLOCK_SIZE;
    int start_tid = start + tid;

    //__shared__ char sub_string_shared [THREAD_BLOCK_SIZE + pattern_length - 1] ;
    __shared__ char sub_string_shared [256] ;

  int pow_2b = 1 << b; 
  unsigned int bit_vector = 0;

  sub_string_shared[tid] = g_input_string[start+tid];

  if ( tid < (pattern_length - 1) ){
     sub_string_shared[THREAD_BLOCK_SIZE + tid] = g_input_string[start+THREAD_BLOCK_SIZE+tid];
  }
  __syncthreads(); 

		

  if (start_tid < (input_size-pattern_length+1)) {

    for (int i = 0; i<pattern_length ; i++) {

        if (sub_string_shared[ tid + i ] == 65) {
            bit_vector = bit_vector + g_pattern_decode[4*i];
        }
        else if (sub_string_shared[ tid + i ] == 67){
            bit_vector = bit_vector + g_pattern_decode[4*i+1];
        }
        else if (sub_string_shared[ tid + i ] == 71){
            bit_vector = bit_vector + g_pattern_decode[4*i+2];
        }
        else if (sub_string_shared[ tid + i ] == 84){   //case of G
            bit_vector = bit_vector + g_pattern_decode[4*i+3];
        }
    }

    //cuPrintf("threadIdx.x = %d \t ,start = %d, bit_vector = %d \n", tid, start, bit_vector);
    //g_matched_result[start] = bit_vector;
    for (int j = no_of_patterns-1; j >= 0; j--) {
        g_matched_result[start_tid*no_of_patterns + j] = bit_vector % pow_2b;
        bit_vector = bit_vector >> b;
    }
  }

}


////////////////////////////////
void ASM_process_top (char *g_input_string, size_t input_size,  int *g_pattern_decode, int pattern_length, int no_of_patterns, int b, int *g_matched_result)
{

    // num_blocks = # of thread blocks to cover input stream
    int num_blocks = (input_size-pattern_length+1)/THREAD_BLOCK_SIZE + 1 ;


        dim3  dimBlock( THREAD_BLOCK_SIZE, 1 ) ;
        dim3  dimGrid ;

        int p = num_blocks >> 15 ;
        dimGrid.x = num_blocks ;
        if ( p ){
            dimGrid.x = 1<<15 ;
            dimGrid.y = p+1 ;
        }
    //cudaPrintfInit();////for cuPrintf

    ASM_kernel <<< dimGrid, dimBlock >>>((char*)g_input_string, input_size, (int*) g_pattern_decode, pattern_length, no_of_patterns, b, g_matched_result);

    //cudaPrintfDisplay();////for cuPrintf
    //cudaPrintfEnd();        ////for cuPrintf
}


int main(int argc, char **argv)
{
   
    char inputFile[FILENAME_MAXLEN];
    char patternFile[FILENAME_MAXLEN];
    strcpy( inputFile, argv[2]) ;
    strcpy( patternFile, argv[1]) ;
    //int k_par = 4;
    int k_par;
    k_par = strtol(argv[3], NULL, 10);
////////////////////////////////////////////////////////////////////////////////////
//Process input patterns
    int pattern_size;
    char *h_pattern = NULL ;
    int len;

    size_t  sizeOfTableEntry ;
    size_t  sizeOfTableInBytes ; // numOfTableEntry * sizeOfTableEntry
    int  sizeOfTableDecodeInBytes ; // numOfTableEntry * sizeOfTableEntry
    size_t  sizeOfPatternInBytes ; // no_of_patterns * sizeOfTableEntry

    int max_pattern_length = 0;
    int no_of_patterns = 0;

    FILE* fpattern = fopen( patternFile, "rb");
    assert ( NULL != fpattern ) ;
    // obtain pattern file
    fseek (fpattern , 0 , SEEK_END);
    pattern_size = ftell (fpattern);
    rewind (fpattern);

    // allocate a buffer to contains all patterns
    h_pattern = (char *) malloc (sizeof(char)*pattern_size);
    assert( NULL != h_pattern );

    // copy the file into the buffer
    pattern_size = fread (h_pattern, 1, pattern_size, fpattern);
    fclose(fpattern);
    //printf ("pattern size = %d\n",pattern_size); 
    //printf ("pattern = %s\n",h_pattern); 
    //Processing to get max_pattern_length & no_of_patterns
    len = 0;
    for( int i = 0 ; i < pattern_size ; i++){
        if ( '\n' == h_pattern[i] ){
            if ( (i > 0) && ('\n' != h_pattern[i-1]) ){ // non-empty line
		no_of_patterns = no_of_patterns + 1;	
		if (max_pattern_length < len+1) {max_pattern_length = len+1;}
            }
            len = 0 ;
        }else{
            len++ ;
        }
    }

   // Create pattern_table, pattern_length_table
    sizeOfTableEntry = sizeof(int) ;
    sizeOfPatternInBytes = no_of_patterns * sizeOfTableEntry; // 1-D to store size of each patterns
    sizeOfTableInBytes = no_of_patterns * max_pattern_length * sizeOfTableEntry; //2-D to store patterns
    sizeOfTableDecodeInBytes = 4 * (max_pattern_length-1); // 1-D to store size of each patterns

    int* pattern_table = (int*) malloc( sizeOfTableInBytes ) ;
    int* pattern_length_table = (int*) malloc( sizeOfPatternInBytes ) ;
    int* h_pattern_decode = (int*) malloc( sizeof(int)*sizeOfTableDecodeInBytes ) ;

    //Processing to fill pattern_table & pattern_length_table 
    len = 0;
    int no_patterns = 0;
    for( int i = 0 ; i < pattern_size ; i++){
        if ( '\n' == h_pattern[i] ){
            if ( (i > 0) && ('\n' != h_pattern[i-1]) ){ // non-empty line
		pattern_length_table[no_patterns] = len;
		no_patterns = no_patterns + 1;	
            }
            len = 0 ;
        }else{
	    pattern_table[no_patterns*max_pattern_length + len] = h_pattern[i]; 	
            len++ ;
        }
    }
   //Print to pattern_table/pattern_length_table to check
/*
    for (int i = 0; i < no_of_patterns; i++) {
	printf("\npattern no %d has length = %d-> ",i, pattern_length_table[i]);
	for (int j = 0; j < pattern_length_table[i]; j++) {
	    printf("%4d",pattern_table[i*max_pattern_length+j]);
	}
    }
*/
//Preprocessing
unsigned int vector_A = 0;
unsigned int vector_C = 0;
unsigned int vector_G = 0;
unsigned int vector_T = 0;
int b = 4;

    for (int i = 0; i< (max_pattern_length-1); i++) {
	vector_A = 0;
	vector_C = 0;
	vector_G = 0;
	vector_T = 0;
	for (int j = 0; j< no_of_patterns; j++) {
	    vector_A = vector_A << b;
	    if (pattern_table[i + max_pattern_length*j] != 65) {vector_A = vector_A + 1;};
	}
	h_pattern_decode[4*i] = vector_A;
		
	for (int j = 0; j< no_of_patterns; j++) {
	    vector_C = vector_C << b;
	    if (pattern_table[i + max_pattern_length*j] != 67) {vector_C = vector_C + 1;};
	}
	h_pattern_decode[4*i+1] = vector_C;

	for (int j = 0; j< no_of_patterns; j++) {
	    vector_G = vector_G << b;
	    if (pattern_table[i + max_pattern_length*j] != 71) {vector_G = vector_G + 1;};
	}
	h_pattern_decode[4*i+2] = vector_G;

	for (int j = 0; j< no_of_patterns; j++) {
	    vector_T = vector_T << b;
	    if (pattern_table[i + max_pattern_length*j] != 84) {vector_T = vector_T + 1;};
	}
	h_pattern_decode[4*i+3] = vector_T;
    }
	
    //for (int i = 0; i < (max_pattern_length-1)*4; i++) {
//	printf("i = %d -> h_pattern_decode = %d\n",i,h_pattern_decode[i]);
    //}
///////////////////////////////////////////////////////////////
//Prepare input string
    int input_size;
    char *h_input_string = NULL ;
    int  *h_matched_result = NULL ;

    //open to read file
    FILE* fpin = fopen( inputFile, "rb");
    assert ( NULL != fpin ) ;
    
    // sets the file position of the stream to the given offset. 
    fseek (fpin , 0 , SEEK_END);
    input_size = ftell (fpin);
    rewind (fpin);
    
    // allocate memory to contain the whole file
    h_input_string = (char *) malloc (sizeof(char)*input_size);
    assert( NULL != h_input_string );

    int size_matched_result = input_size * no_of_patterns;
    //int size_matched_result = input_size;
    h_matched_result = (int *) malloc (sizeof(int)*size_matched_result); // each input has no_of_patterns results
    assert( NULL != h_matched_result );
    memset( h_matched_result, 0, size_matched_result ) ;

    // copy the file into the buffer
    input_size = fread (h_input_string, 1, input_size, fpin);
    fclose(fpin);

    //////////////////
    //printf("\ninput size -> %4d -> \n",input_size);
   // printf("%s\n",h_input_string);

    //AmSM with Proposed Method in CPU
/*    
int vector = 0;
    struct timespec t_start, t_end;
    double elapsedTime;
    clock_gettime (CLOCK_REALTIME, &t_start);
    //printf ("starttime s = %li, ns = %li\n",t_start.tv_sec, t_start.tv_nsec);
    for(int i = 0; i < input_size-maxpattern_length+1; i++) {
	vector = 0;
	for (int k = 0; k < max_pattern_length-1; k++) {
	     if (h_input_string[i+k] == 65) {
		vector = vector + h_pattern_decode[4*k];
	     }
	     else if (h_input_string[i+k] == 67) {
		vector = vector + h_pattern_decode[4*k+1];
	     }
	     else if (h_input_string[i+k] == 71) {
		vector = vector + h_pattern_decode[4*k+2];
	     }
	     else if (h_input_string[i+k] == 84) {
		vector = vector + h_pattern_decode[4*k+3];
	     }
	}
    //printf("vector = %d\n",vector);
	for (int j = no_of_patterns-1; j >= 0; j--) { 
	     h_matched_result[i*no_of_patterns+j] = vector % 64;
	     vector = vector >> b;
        }
    }
    clock_gettime(CLOCK_REALTIME, &t_end);
    //printf ("endtime s = %li, ns = %li\n",t_end.tv_sec, t_end.tv_nsec);
    elapsedTime = (t_end.tv_sec*1000+t_end.tv_nsec/1000000)-(t_start.tv_sec*1000+t_start.tv_nsec/1000000);
*/


    //AmSM with Proposed Method in GPU
    char *g_input_string;
    //char *g_pattern;
    int *g_matched_result;
    int *g_pattern_decode;

    hipMalloc (&g_input_string, sizeof(char)*input_size);
    hipMalloc (&g_matched_result, sizeof(int)*size_matched_result);
    hipMalloc (&g_pattern_decode, sizeof(int)*sizeOfTableDecodeInBytes);

    hipMemcpy (g_input_string, h_input_string, sizeof(char)*input_size, hipMemcpyHostToDevice );
    hipMemcpy (g_pattern_decode, h_pattern_decode, sizeof(int)*sizeOfTableDecodeInBytes, hipMemcpyHostToDevice);

    // record time setting
    hipEvent_t start, stop;
    float time;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);


    // step 3: run ASM on GPU           
    ASM_process_top ( g_input_string, input_size, g_pattern_decode, (max_pattern_length-1), no_of_patterns, b,g_matched_result) ;

    // record time setting
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);

    hipMemcpy (h_matched_result, g_matched_result, sizeof(int)*size_matched_result, hipMemcpyDeviceToHost );

    // Print Result
int total_result = 0;
    for(int i = 0; i < (input_size-max_pattern_length+1)*no_of_patterns; i++) {
	     //printf("Input location %d has Hamming distance = %d\n",i, h_matched_result[i]);
	     if(h_matched_result[i] <= k_par) {total_result++;}
        }
    printf("\n\n\n");
    printf("###########################################################\n");
    printf("#--Multi Fix-Length Patterns Approximate String Matching--#\n");
    printf("#---------------------------------------------------------#\n");
    printf("#------------ Shared-Memory-based PMASM ------------------#\n");
    printf("###########################################################\n");
    printf("#--No of Patterns            |\t\t %10d \t  #\n",no_of_patterns);
    printf("#---------------------------------------------------------#\n");
    printf("#--Pattern Length            |\t\t %10d \t  #\n",max_pattern_length-1);
    printf("#---------------------------------------------------------#\n");
    printf("#--Input Size (bytes)        |\t\t %10d \t  #\n", input_size );
    printf("#---------------------------------------------------------#\n");
    printf("#--Total matched with k = %d  |\t\t %10d \t  #\n", k_par, total_result);
    printf("#---------------------------------------------------------#\n");
    printf("#--Total elapsed time (ms)   |\t\t %10f \t  #\n", time);
    printf("#---------------------------------------------------------#\n");
    printf("#--Throughput Result (Gbps)  |\t\t %10f \t  #\n", (float)(input_size*8)/(time*1000000) );
    printf("###########################################################\n");

     
    free(h_pattern);
    free(h_input_string);
    free(h_matched_result); 
    free(pattern_table);
    free(pattern_length_table);
    free(h_pattern_decode);
    hipFree(g_input_string);
    hipFree(g_pattern_decode);
    hipFree(g_matched_result);
            
    return 0;
}
